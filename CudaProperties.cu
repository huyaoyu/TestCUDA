#include <cmath>
#include <iostream>

int main(void)
{
    int devices = 0;

    hipGetDeviceCount(&devices);

    hipDeviceProp_t prop;

    for ( int i = 0; i < devices; ++i )
    {
        hipGetDeviceProperties(&prop, i);

        std::cout << "=== Device number " << i << " ===" << std::endl;
        std::cout << "name = " << prop.name << std::endl;
        std::cout << "totalGlobalMem = " << prop.totalGlobalMem << std::endl;
        std::cout << "totalGlobalMem (MB) = " << prop.totalGlobalMem / 1024.0 / 1024 << std::endl;
        std::cout << "sharedMemPerBlock = " << prop.sharedMemPerBlock << std::endl;
        std::cout << "sharedMemPerBlock (kB) = " << prop.sharedMemPerBlock / 1024.0 << std::endl;
        std::cout << "regsPerBlock = " << prop.regsPerBlock << std::endl;
        std::cout << "warpSize = " << prop.warpSize << std::endl;
        std::cout << "memPitch = " << prop.memPitch << std::endl;
        std::cout << "maxThreadsPerBlock = " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "maxThreadsDim[3] = [" 
                  << prop.maxThreadsDim[0] << ", "
                  << prop.maxThreadsDim[1] << ", "
                  << prop.maxThreadsDim[2] 
                  << "]" << std::endl;
        std::cout << "maxGridSize[3] = [" 
                  << prop.maxGridSize[0] << ", "
                  << prop.maxGridSize[1] << ", "
                  << prop.maxGridSize[2]
                  << "]" << std::endl;
        std::cout << "totalConstMem = " << prop.totalConstMem << std::endl;
        std::cout << "major = " << prop.major << std::endl;
        std::cout << "minor = " << prop.minor << std::endl;
        std::cout << "clockRate = " << prop.clockRate << std::endl;
        std::cout << "textureAlignment = " << prop.textureAlignment << std::endl;
        std::cout << "deviceOverlap = " << prop.deviceOverlap << std::endl;
        std::cout << "multiProcessorCount = " << prop.multiProcessorCount << std::endl;
        std::cout << "kernelExecTimeoutEnabled = " << prop.kernelExecTimeoutEnabled << std::endl;
        std::cout << "integrated = " << prop.integrated << std::endl;
        std::cout << "canMapHostMemory = " << prop.canMapHostMemory << std::endl;
        std::cout << "computeMode = " << prop.computeMode << std::endl;
        std::cout << "concurrentKernels = " << prop.concurrentKernels << std::endl;
        std::cout << "ECCEnabled = " << prop.ECCEnabled << std::endl;
        std::cout << "pciBusID = " << prop.pciBusID << std::endl;
        std::cout << "pciDeviceID = " << prop.pciDeviceID << std::endl;
        std::cout << "tccDriver = " << prop.tccDriver << std::endl; 
        std::cout << std::endl;
    }

    return 0;
}
